#include "hip/hip_runtime.h"
#include "BilateralFilter.h"

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg, file_name, line_number, hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define CHECK_CUDA_ERROR(call, msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

// Kernel code for bilateral filtering of color image
__global__ 
void color_bilateral_filter(const float* input,
							const float* kernel,
							const float r,
							const int w,
							const int width, 
							const int height,
							const int step,
							float* output)
{
	// 2D Index of current thread
	const int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

	//// Only valid threads can perform memory I/O
	if ((xIdx < width) && (yIdx < height))
	{
		const char* inCharPtr = (char*)input;
		const float* srcRow = (const float*)(inCharPtr + yIdx*step);
		float L = srcRow[xIdx];
		float a = srcRow[xIdx + 1];
		float b = srcRow[xIdx + 2];

		for(int i = -w; i <= w; i++)
			for (int j = -w; j <= w; j++)
			{
				int x_sample = xIdx + i;
				int y_sample = yIdx + j;

				// mirror edges
				if (x_sample < 0) x_sample = -x_sample;
				if (y_sample < 0) y_sample = -y_sample;
				if (x_sample > width - 1) x_sample = width - 1 - i;
				if (y_sample > height - 1) y_sample = height - 1 - i;


			}
	}
}

// Kernel code for bilateral filtering of gray image
__global__ 
void gray_bilateral_filter(const float* input,
							const float* kernel,
							const float r,
							const int w,
							const int width,
							const int height,
							const int step, 
							float* output)
{
	// 2D Index of current thread
	const int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

	// Only valid threads can perform memory I/O
	if ((xIdx < width) && (yIdx < height))
	{

	}
}

Mat BilateralFilter::ApplyFilterCUDA(Mat img)
{
#ifdef _DEBUG
	std::cout << "Data type of image:" << type2str(img.type()) << endl;
	std::cout << "Data type of G Mat:" << type2str(G.type()) << endl;
	cv::Point3_<float>* p = img.ptr<cv::Point3_<float>>(1023, 1023);
	std::cout << "image at (0,0):" << endl <<
		"\t" << p->x << endl <<
		"\t" << p->y << endl <<
		"\t" << p->z << endl;
#endif

	Mat out;

	const size_t bytes = img.step * img.rows;

	const size_t Gbytes = G.step * G.rows;

	float* d_input, *d_output;

	// Allocation of device memory
	CHECK_CUDA_ERROR(hipMalloc<float>(&d_input, bytes), "CUDA Malloc Failed");
	CHECK_CUDA_ERROR(hipMalloc<float>(&d_output, bytes), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	CHECK_CUDA_ERROR(hipMemcpy(d_input, img.ptr<float>(), bytes, hipMemcpyHostToDevice), "CUDA Memcpy Host to Device Failed");

	// Allocate kernel on device memory and copy G to it if it has not already
	if (!d_kernel)
	{
		CHECK_CUDA_ERROR(hipMalloc<float>(&d_kernel, Gbytes), "CUDA Malloc Failed");
		// Copy data from OpenCV Mat G to device memory
		CHECK_CUDA_ERROR(hipMemcpy(d_kernel, G.ptr<float>(), Gbytes, hipMemcpyHostToDevice), "CUDA Memcpy Host to Device Failed");
	}

	// Define block size
	const dim3 block(16, 16);

	// Grid size in order to cover whole image
	const dim3 grid((img.cols + block.x - 1) / block.x, (img.rows + block.y - 1) / block.y);

	if (img.channels() > 1)
	{
		out = Mat::zeros(img.rows, img.cols, CV_32FC3);
		// Launch bilateral filter kernel for color image
		color_bilateral_filter << <grid, block >> >(d_input, d_kernel, r, w, img.cols, img.rows, img.step, d_output);
	}
	else
	{
		out = Mat::zeros(img.rows, img.cols, CV_32FC1);
		// Launch bilateral filter kernel for grayscale image
		gray_bilateral_filter << <grid, block >> >(d_input, d_kernel, r, w, img.cols, img.rows, img.step, d_output);
	}

	// Synchronize to check for kernel launch errors
	CHECK_CUDA_ERROR(hipDeviceSynchronize(), "Kernel Launch Failed");

	// Copy back data from the destination device memory to the OpenCV output image
	CHECK_CUDA_ERROR(hipMemcpy(out.ptr<float>(), d_output, bytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	// Free the device memory
	CHECK_CUDA_ERROR(hipFree(d_input), "CUDA Free Failed");
	CHECK_CUDA_ERROR(hipFree(d_output), "CUDA Free Failed");

	return out;
}

BilateralFilter::~BilateralFilter()
{
	// Free kernel memory
	G.deallocate();

	// If the kernel had its info copied to device, free it
	if(d_kernel)
		CHECK_CUDA_ERROR(hipFree(d_kernel), "CUDA Free Failed");
}